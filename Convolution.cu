#include "hip/hip_runtime.h"
#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

using namespace std;
/**
 * CUDA Kernel Device code
 *
 * Computes the max_pooling of A into C. 
 */

__global__ void Convolution(const float *A, const float *K, float *C, int in_rows, int in_cols, int out_rows, int out_cols, int stride, int kernel_size, int padding)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

	// policy = 0: same
	// policy = 1: valid
	int valid_offset = (padding == 1) ? kernel_size / 2 : 0;
	
	if (i < out_rows && j < out_cols){
        int offset_i = i * stride;
		int offset_j = j * stride;
		float sum = 0;

		for(int k_row_index = -kernel_size/2; k_row_index <= kernel_size/2; k_row_index++){
			for(int k_col_index = -kernel_size/2; k_col_index <= kernel_size/2; k_col_index++){
				int in_row_index =  offset_i + k_row_index + valid_offset;
				int in_col_index =  offset_j + k_col_index + valid_offset;
				if(in_row_index < 0 || in_row_index >= in_rows || in_col_index < 0 || in_col_index >= in_cols)
					continue; // only happens for same mode, skipping is equivalent to summing with padded zeros
				sum += A[in_row_index * in_cols + in_col_index] * K[(k_row_index + kernel_size/2)*kernel_size + (k_col_index + kernel_size/2)];
			}
		}
		C[i * out_cols + j] = sum;
		
	}

}

void HelperOutputDim(int in_row, int in_col, int stride, int kernel_size, int &out_row, int &out_col, int padding_policy){
	
	switch(padding_policy){
		case 0: // any / same
			out_row = (int)ceil((float)in_row / (float)stride);
			out_col = (int)ceil((float)in_col / (float)stride);
			break;
		case 1: // valid
			out_row = (int)ceil((float)(in_row - kernel_size + 1) / (float)stride);
			out_col = (int)ceil((float)(in_col - kernel_size + 1) / (float)stride);
			break;
		default:
			break;
	}
}

int main(void)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
	int in_rows = 5;
	int in_cols = 5;
	int kernel_size = 3;
	int stride = 1;
	int padding_policy = 1;
	size_t size_K = kernel_size * kernel_size* sizeof(float);
    size_t size_A = in_rows * in_cols* sizeof(float);
    printf("[Matrix of %d elements]\n", in_rows * in_cols);

    // Allocate the host input vector A
    float *h_A = (float *)malloc(size_A);

	// Allocate the host input vector K
    float *h_K = (float *)malloc(size_K);

    // Allocate the host output vector C
	int out_rows = 0;
	int out_cols = 0;
	HelperOutputDim(in_rows, in_cols, stride, kernel_size, out_rows, out_cols, padding_policy);
	size_t size_C = out_rows * out_cols* sizeof(float);
    float *h_C = (float *)malloc(size_C);

    // Verify that allocations succeeded
    if (h_A == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
	for(int i = 0; i < in_rows; i++){
		for(int j = 0; j < in_cols; j++){
			h_A[i * in_cols + j] = 1; //i * in_cols + j;
		}
	}
	 // Initialize the host input vectors
	for(int i = 0; i < kernel_size; i++){
		for(int j = 0; j < kernel_size; j++){
			h_K[i * kernel_size + j] = 2;
		}
	}

    // Allocate the device input vector A
    float *d_A = NULL;
    err = hipMalloc((void **)&d_A, size_A);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	// Allocate the device input vector K
    float *d_K = NULL;
    err = hipMalloc((void **)&d_K, size_K);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector K (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
  
    // Allocate the device output vector C
    float *d_C = NULL;
    err = hipMalloc((void **)&d_C, size_C);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors A and K in host memory to the device input vectors in
    // device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	 // device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_K, h_K, size_K, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector K from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Launch the MaxPooling CUDA Kernel
	dim3 threadsPerBlock(16, 16);
    dim3 numBlocks(in_rows / threadsPerBlock.x + 1, in_cols / threadsPerBlock.y + 1);
    Convolution<<<numBlocks, threadsPerBlock>>>(d_A, d_K, d_C, in_rows, in_cols, out_rows, out_cols, stride, kernel_size, padding_policy);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Verify that the result vector is correct
    
	for(int i = 0; i < out_rows; i++){
		for(int j = 0; j < out_cols; j++){
			printf("%f ",h_C[i*out_cols+j]);
		}
		printf("\n");
	}

    printf("Test PASSED\n");

    // Free device global memory
    err = hipFree(d_A);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	err = hipFree(d_K);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector K (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_C);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_A);
	free(h_K);
    free(h_C);

    printf("Done\n");
    return 0;
}
